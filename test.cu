/* Copyright (C) 2022 Barcelona Supercomputing Center
 * SPDX-License-Identifier: MIT
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include<math.h>
#include<iostream>
#include<hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

const int N = 16;
const int blocksize = 16;


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__
void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("HANDLE_ERROR %s", a);

	HANDLE_ERROR(hipMalloc( (void**)&ad, csize ));
	//cudaMalloc( (void**)&ad, csize );
	
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );

	printf("%s\n", a);
	return EXIT_SUCCESS;
}
