#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

#include "libsolv.h"

//#include<hipblas.h>
//#include<hipblas.h>

using namespace std;

//
//dAthreads
//
// Para reservar memoria Double e Int
extern "C++" void cudaMallocDouble(double* &vector,int size)
{
	hipMalloc((void**)&vector,size*sizeof(double));
}

extern "C++" void cudaMallocInt(int* &vector,int size)
{
	hipMalloc((void**)&vector,size*sizeof(int));
}

// Para copiar a CPU->GPU Double e Int
extern "C++" void cudaMemcpyDToGpu(double* h_vect,double* d_vect,int size )
{
  hipMemcpy(d_vect,h_vect,size*sizeof(double),hipMemcpyHostToDevice);
}

extern "C++" void cudaMemcpyIToGpu(int* h_vect,int* d_vect,int size )
{
		hipMemcpy(d_vect,h_vect,size*sizeof(int),hipMemcpyHostToDevice);
}

// Para copiar a GPU->CPU Double e Int
extern "C++" void cudaMemcpyIToCpu(int* h_vect, int* d_vect,int size )
{
		hipMemcpy(h_vect,d_vect,size*sizeof(int),hipMemcpyDeviceToHost);
}

extern "C++" void cudaMemcpyDToCpu(double* h_vect, double* d_vect,int size )
{
  hipMemcpy(h_vect,d_vect,size*sizeof(double),hipMemcpyDeviceToHost);
}

// Para liberar memoria
extern "C++" void cudaFreeMem(void* vector)
{
	hipFree(vector);
}

extern "C++" void cudaGetLastErrorC(){
     hipError_t error;
     error=hipGetLastError();
     if(error!= hipSuccess)
     {
       cout<<" ERROR INSIDE A CUDA FUNCTION: "<<error<<" "<<hipGetErrorString(error)<<endl;
       exit(0);
     }
}

__global__ void cudamatScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      if(djA[j]==row)
      {
        dA[j] = 1.0 + alpha*dA[j];
      }
      else{
        dA[j] = alpha*dA[j];
      }
    }
  }
}

// A = I - gamma*J
// dA  : Matrix values (nnz size)
// djA : Matrix columns (nnz size)
// diA : Matrix rows (nrows+1 size)
// alpha : Scale factor
extern "C++" void gpu_matScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha, int blocks, int threads)
{

   blocks = (nrows+threads-1)/threads;

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

  cudamatScaleAddI<<<dimGrid,dimBlock>>>(nrows, dA, djA, diA, alpha);
}

__global__
void check_input_gpud(double *x, int len, int var_id)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  printf("%d[%d]=%-le\n",var_id,i,x[i]);

}

// Diagonal precond
__global__ void cudadiagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;

#ifdef DEBUG_cudadiagprecond


#endif

  if(row < nrows){
    int jstart=diA[row];
    int jend  =diA[row+1];
    for(int j=jstart;j<jend;j++){
      if(djA[j]==row){
        if(dA[j]!=0.0)
          ddiag[row]= 1.0/dA[j];
        else{
          //printf("cudadiagprecond else\n");
          ddiag[row]= 1.0;
        }
      }
    }
  }

}

extern "C++" void gpu_diagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag, int blocks, int threads)
{

  blocks = (nrows+threads-1)/threads;

  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudadiagprecond<<<dimGrid,dimBlock>>>(nrows, dA, djA, diA, ddiag);
  //check_input_gpud<< < 1, 5>> >(ddiag,nrows,0);
}

// y = constant
__global__ void cudasetconst(double* dy,double constant,int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=constant;
	}
}

extern "C++" void gpu_yequalsconst(double *dy, double constant, int nrows, int blocks, int threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudasetconst<<<dimGrid,dimBlock>>>(dy,constant,nrows);

}


// x=A*b
__global__ void cudaSpmvCSR(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    double sum = 0.0;
    for(int j=jstart; j<jend; j++)
    {
      sum+= db[djA[j]]*dA[j];
    }
    dx[row]=sum;
	}

}

__global__ void cudaSpmvCSC(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
	double mult;
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      mult = db[row]*dA[j];
      atomicAdd(&(dx[djA[j]]),mult);
    }
	}
}

extern "C++" void gpu_spmv(double* dx ,double* db, int nrows, double* dA, int *djA,int *diA,int mattype,int blocks,int  threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   if(mattype==0)
   {
     cudaSpmvCSR<<<dimGrid,dimBlock>>>(dx, db, nrows, dA, djA, diA);
   }
   else
   {
	    cudasetconst<<<dimGrid,dimBlock>>>(dx, 0.0, nrows);
	    cudaSpmvCSC<<<dimGrid,dimBlock>>>(dx, db, nrows, dA, djA, diA);
   }
}

// y= a*x+ b*y
__global__ void cudaaxpby(double* dy,double* dx, double a, double b, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]= a*dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_axpby(double* dy ,double* dx, double a, double b, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudaaxpby<<<dimGrid,dimBlock>>>(dy,dx,a,b,nrows);
}

// y = x
__global__ void cudayequalsx(double* dy,double* dx,int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=dx[row];
	}
}

extern "C++" void gpu_yequalsx(double *dy, double* dx, int nrows, int blocks, int threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudayequalsx<<<dimGrid,dimBlock>>>(dy,dx,nrows);

}

__global__ void cudareducey(double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;

  double mySum =  (tid < n) ? g_odata[tid] : 0;

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void cudadotxy(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;//*2 because init blocks is half
  //unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;//*2 because init blocks is half

  double mySum = (i < n) ? g_idata1[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  //for (unsigned int s=(blockDim.x+1)/2; s>0; s>>=1)
  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

//threads need to be pow of 2 //todo h_temp not needed
extern "C++" double gpu_dotxy(double* vec1, double* vec2, double* h_temp, double* d_temp, int nrows, int blocks,int threads)
{
  double sum;
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  //threads*sizeof(double)
  cudadotxy<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,nrows);
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);
  //printf("rho1 %f", sum);

  int redsize= sqrt(blocks) +1;
  redsize=pow(2,redsize);

  dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(redsize,1,1);

  cudareducey<<<dimGrid2,dimBlock2,redsize*sizeof(double)>>>(d_temp,blocks);
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);

  return sum;

/*
  hipMemcpy(h_temp, d_temp, blocks * sizeof(double), hipMemcpyDeviceToHost);
  double sum=0;
  for(int i=0;i<blocks;i++)
  {
    sum+=h_temp[i];
  }
  return sum;
*/
  /*dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(blocks,1,1);

  //Cuda only sum kernel call
  //cudareducey<<<dimGrid2,dimBlock2,blocks*sizeof(double)>>>(d_temp,blocks); //Takes quasi WAY MORE than cpu calc

  hipMemcpy(h_temp, d_temp, sizeof(double), hipMemcpyDeviceToHost);
  return h_temp[0];*/
}

/*
extern "C++" double gpu_dotxy(double *dy, double* dx, int nrows)
{
   double dot=0.0;
   hipblasHandle_t hl;
   hipblasCreate(&hl);

   hipblasDdot(hl,nrows,dy,1,dx,1,&dot);

   hipblasDestroy(hl);
   return dot;
}
*/

// z= a*z + x + b*y
__global__ void cudazaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=a*dz[row]  + dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_zaxpbypc(double* dz, double* dx ,double* dy, double a, double b, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudazaxpbypc<<<dimGrid,dimBlock>>>(dz,dx,dy,a,b,nrows);
}

// z= x*y
__global__ void cudamultxy(double* dz, double* dx,double* dy, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=dx[row]*dy[row];
	}
}

extern "C++" void gpu_multxy(double* dz, double* dx ,double* dy, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudamultxy<<<dimGrid,dimBlock>>>(dz,dx,dy,nrows);
}

// a*x + b*y = z
//__global__ void cudazaxpby(double* dz, double* dx,double* dy, double a, double b, int nrows)
__global__ void cudazaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=a*dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_zaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

  cudazaxpby<<<dimGrid,dimBlock>>>(a,dx,b,dy,dz,nrows);
}

// y= a*x + y
__global__ void cudaaxpy(double* dy,double* dx, double a, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=a*dx[row] + dy[row];
	}
}

extern "C++" void gpu_axpy(double* dy, double* dx ,double a, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudaaxpy<<<dimGrid,dimBlock>>>(dy,dx,a,nrows);
}

// sqrt(sum ( (x_i*y_i)^2)/n)
__global__ void cudaDVWRMS_Norm(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  double mySum = (i < n) ? g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C++" double gpu_VWRMS_Norm(int n, double* vec1,double* vec2,double* h_temp,double* d_temp, int blocks,int threads)
{
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudaDVWRMS_Norm<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,n);

  //hipMemcpy(h_temp, d_temp, blocks * sizeof(double), hipMemcpyDeviceToHost);

  int redsize= sqrt(blocks) +1;
  redsize=pow(2,redsize);

  dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(redsize,1,1);

  cudareducey<<<dimGrid2,dimBlock2,redsize*sizeof(double)>>>(d_temp,blocks);

  double sum;
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);

  return sqrt(sum/n);

/*
  double sum=0;
  for(int i=0;i<blocks;i++)
  {
    sum+=h_temp[i];
  }
  return sqrt(sum/n);
  */
}

// y=alpha*y
__global__ void cudascaley(double* dy, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=a*dy[row];
  }
}

extern "C++" void gpu_scaley(double* dy, double a, int nrows, int blocks, int threads)
{
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudascaley<<<dimGrid,dimBlock>>>(dy,a,nrows);
}




// Device functions (equivalent to global functions but in device to allow calls from gpu)
__device__ void cudaDevicematScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      if(djA[j]==row)
      {
        dA[j] = 1.0 + alpha*dA[j];
      }
      else{
        dA[j] = alpha*dA[j];
      }
    }
}

// Diagonal precond
__device__ void cudaDevicediagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  int jstart=diA[row];
  int jend  =diA[row+1];
  for(int j=jstart;j<jend;j++){
    if(djA[j]==row){
      if(dA[j]!=0.0)
        ddiag[row]= 1.0/dA[j];
      else{
        ddiag[row]= 1.0;
      }
    }
  }

}

// y = constant
__device__ void cudaDevicesetconst(double* dy,double constant,int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=constant;
}

// x=A*b
__device__ void cudaDeviceSpmvCSR(double* dx, double* db, double* dA, int* djA, int* diA)
{
  __syncthreads();
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  int tid=threadIdx.x;
  double sum = 0.0;
  int nnz=diA[blockDim.x];

#ifdef CSR_SHARED

  extern __shared__ double sdata[];

  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++)
  {
    //sdata[j]=dA[j+nnz*blockIdx.x]; //slower than storing earlier (just after the last reduce operation, which is the operation that uses shared last time)
    sum+= db[djA[j]+blockDim.x*blockIdx.x]*sdata[j];
  }
  dx[row]=sum;
  __syncthreads();

#elif CSR_SHARED_DB

  extern __shared__ double sdata[];
  sdata[tid] = db[row];
  __syncthreads();

  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++)
  {
    sum+= sdata[djA[j]]*dA[j+nnz*blockIdx.x];
  }
  dx[row]=sum;
  __syncthreads();

#elif CSR_SHARED_DB_JAC

  extern __shared__ double sdata[];
  sdata[nnz+tid] = db[row];
  __syncthreads();

  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++)
  {
    sum+= sdata[nnz+djA[j]]*sdata[j];
  }
  dx[row]=sum;
  __syncthreads();

#else

  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++)
  {
    sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
  }
  dx[row]=sum;
  __syncthreads();

#endif

}

__device__ void cudaDeviceaddD(double *g_odata, double in, volatile double *sdata, int n_shr_empty)
{
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();

  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];

  __syncthreads();

  //if(blockIdx.x==0)printf("i %d in %le sdata[tid] %le\n",i,in,sdata[tid]);

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){//&& sdata[tid + s]!=0.
      //if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void cudaDeviceSpmvCSCAtomic(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty)
{
  double mult;
  extern __shared__ double sdata[];
  int row= threadIdx.x + blockDim.x*blockIdx.x;

  __syncthreads();
  dx[row]=0.0;
  __syncthreads(); //Multiple threads can save to the same row

  int nnz=diA[blockDim.x];
    for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++)
    {

#ifdef DEBUG_CUDADEVICESPMVCSC_BLOCK
      if(dA[j] != dA[j])
        printf("NAN dA[j]");
      if(djA[j] != djA[j])
        printf("NAN djA[j]]");
#endif

      mult = db[row]*dA[j+nnz*blockIdx.x];
      //atomicAdd(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
      atomicAdd_block(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
//		dx[djA[j]]+= db[row]*dA[j];
    }

  __syncthreads();
}

__device__ void cudaDeviceSpmvCSD(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[tid]=0.0;
  __syncthreads();

  //int iRow=threadIdx.x;
  //dx[tid]+=db[tid]*dA[tid]; //main diagonal
  //iDx++;
  int nnz=1118;//todo
  for(int iDiag=0; iDiag<blockDim.x; iDiag++) {

    if(threadIdx.x<diA[iDiag+1]-diA[iDiag]) {
      int dAi=diA[iDiag] + threadIdx.x  + nnz * blockIdx.x;
      int dbi=djA[diA[iDiag] + threadIdx.x] + blockDim.x*blockIdx.x;
      int dxi=((iDiag+djA[diA[iDiag] + threadIdx.x])%blockDim.x) + blockDim.x*blockIdx.x;
      dx[dxi] += db[dbi] * dA[dAi];
    }

    __syncthreads();
  }

}

__device__ void cudaDeviceSpmvBoolDet(double* dx, double* db, double* dA, int* diA)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[tid]=0.0;
  __syncthreads();

  //dx[tid]+=db[tid]*dA[tid]; //main diagonal

  /*

  int id=tid;
  dx[tid]+=db[tid]*dA[tid];
  for(int i=1; i<nrows; i++) {
    id+=i;
    if(id==nrows){
      id=0;
    }
    if(diA[i*nrows+tid]==1)
      dx[id]+=db[tid]*dA[tid+i*nrows];


    __syncthreads();
  }
*/

}

__device__ void cudaDeviceSpmvCUID(double* dx, double* db, double* dA, int* djA)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[tid]=0.0;
  __syncthreads();

  //dx[tid]+=db[tid]*dA[tid]; //main diagonal

  //[n_row*n_row] with the index to access A for each tid
  //e.g. [0 -1  1]  Thread 0 access index 1 of A, Thread 1 skips because there is no number here (zero in original matrix), Thread 2 access  A[iRow*n_row+1]

  //if(tid==0)printf("cudaDeviceSpmvCUID\n");

  int nnz=1118;//todo
  int iRow=threadIdx.x;
  //dx[tid]+=db[tid]*dA[tid];
  //iRow++;
  __syncthreads();
  for(int row=0; row<blockDim.x; row++) {
    if (djA[threadIdx.x + row * blockDim.x] >= 0) {
      dx[iRow + blockDim.x * blockIdx.x] +=
          db[tid] * dA[djA[threadIdx.x + row * blockDim.x] + nnz * blockIdx.x];
#ifdef DEBUG_CUID
      printf("dx db dA djA %lf %lf %d\n", dx[iRow], db[tid],dA[djA[tid + row * blockDim.x]], djA[tid + row * blockDim.x]);
#endif
    }
    iRow++;
    if (iRow >= blockDim.x) {
      iRow = 0;
    }
      __syncthreads();
    }

}

__device__ void cudaDeviceSpmvCSRVector(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty)
{
  // Thread ID in block
  int t = threadIdx.x;

  int warpSize = 32;

  // Thread ID in warp
  int lane = t & (warpSize-1);

  // Number of warps per block
  int warpsPerBlock = blockDim.x / warpSize;

  // One row per warp
  int row = (blockIdx.x * warpsPerBlock) + (t / warpSize);

  extern __shared__ double vals[];
  //__shared__ volatile double vals[n_shr_empty+blockDim.x];

  unsigned int tid = threadIdx.x;
  if(tid<n_shr_empty)
    vals[tid+blockDim.x]=0.;

  int rowStart = diA[row];
  int rowEnd = diA[row+1];
  double sum = 0.;

  // Use all threads in a warp accumulate multiplied elements
  for (int j = rowStart + lane; j < rowEnd; j += warpSize)
  {
    int col = djA[j];
    sum += dA[j] * db[col];
  }
  vals[t] = sum;
  __syncthreads();

  // Reduce partial sums
  if (lane < 16) vals[t] += vals[t + 16];
  if (lane <  8) vals[t] += vals[t + 8];
  if (lane <  4) vals[t] += vals[t + 4];
  if (lane <  2) vals[t] += vals[t + 2];
  if (lane <  1) vals[t] += vals[t + 1];
  __syncthreads();

  // Write result
  if (lane == 0)
  {
    dx[row] = vals[t];
  }

}

#ifdef CSR_ADAPTIVE

int nextPowerOfTwo(int v) {

  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;

  return v;
}

__device__ void cudaDevicedotxyCSRReduce(double *g_idata, double *g_idata2,
                                double *g_odata, int n, int n_shr_empty, int n_shr_len)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();
  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;

  __syncthreads();
  sdata[tid] = g_idata[i]*g_idata2[i];
  __syncthreads();

    for (unsigned int s=(n_shr_len)/2; s>0; s>>=1)
    {
      if (tid < s)
        sdata[tid] += sdata[tid + s];
      __syncthreads();
    }
    *g_odata = sdata[0];
    __syncthreads();
}

__device__ void cudaDeviceSpmvCSRReduce(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
  __syncthreads();
  int row = threadIdx.x + blockDim.x*blockIdx.x;
  double sum = 0.0;

  int nnz=diA[blockDim.x];
  //for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++)
  int n_iters = nnz / blockDim.x; //todo /2?
  for(int i=0; i<n_iters; i++)
  {



    int offsetdA=diA[threadIdx.x+1]-diA[threadIdx.x];
    int n_shr_len=nextPowerOfTwo(offsetdA);
    int n_shr_empty=n_shr_len-(offsetdA);
    int j=row;
    dx[row] = db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
    int idx=threadIdx.x/offsetdA;
    cudaDevicedotxyCSRReduce(&db[djA[j]+blockDim.x*blockIdx.x],
                             &dA[j+nnz*blockIdx.x],&dx[idx],n_shr_empty,n_shr_len);


    //sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
  }
  //dx[row]=sum;
  __syncthreads();

  int residual=nnz-(blockDim.x*n_iters);
  if(threadIdx.x < residual){


  }

}

#endif

__device__ void cudaDeviceSpmv(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty)
{

#ifdef CSR
  cudaDeviceSpmvCSR(dx,db,dA,djA,diA);
#elif CSC_ATOMIC
  cudaDeviceSpmvCSCAtomic(dx,db,dA,djA,diA,n_shr_empty);
#elif CSD
  cudaDeviceSpmvCSD(dx,db,dA,djA,diA,n_shr_empty);
#elif CBD
  cudaDeviceSpmvBoolDet(dx,db,dA,djA);
#elif CUID
  cudaDeviceSpmvCUID(dx,db,dA,djA);
#elif CSR_VECTOR
  cudaDeviceSpmvCSRVector(dx,db,dA,djA,diA,n_shr_empty);
#elif CSR_ADAPTIVE
  cudaDeviceSpmvCSRReduce(dx,db,dA,djA,diA,n_shr_empty);
#else
  cudaDeviceSpmvCSR(dx,db,dA,djA,diA);
#endif

}

// y= a*x+ b*y
__device__ void cudaDeviceaxpby(double* dy,double* dx, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]= a*dx[row] + b*dy[row];
}

// y = x
__device__ void cudaDeviceyequalsx(double* dy,double* dx,int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
    dy[row]=dx[row];
}

__device__ void cudaDevicemin(double *g_odata, double in, volatile double *sdata, int n_shr_empty)
{
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();
  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads(); //Not needed (should)

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void cudaDevicemaxI(int *g_odata, int in, volatile double *sdata, int n_shr_empty)
{
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();
  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads(); //Not needed (should)

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] > sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void cudaDeviceaddI(int *g_odata, int in, volatile double *sdata, int n_shr_empty)
{
  //extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();

  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];

  __syncthreads(); //Not needed (should)

  //if(blockIdx.x==0)printf("i %d in %le sdata[tid] %le\n",i,in,sdata[tid]);

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){//&& sdata[tid + s]!=0.
      //if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
  unsigned int blockSize = blockDim.x;
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}


__device__ void cudaDevicedotxy(double *g_idata1, double *g_idata2,
                                 double *g_odata, int n_shr_empty)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  //Needed, when testing be careful with SRAM data remanesce https://stackoverflow.com/questions/22172881/why-does-my-kernels-shared-memory-seems-to-be-initialized-to-zero

  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;

  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata2[i];
  __syncthreads();

/*
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  */

  //todo treat case deriv_length < 32
  //maybe https://github.com/cudpp/cudpp/blob/master/src/cudpp/kernel/reduce_kernel.cuh


  unsigned int blockSize = blockDim.x+n_shr_empty;

  // do reduction in shared mem
  if ((blockSize >= 1024) && (tid < 512)) {
    sdata[tid] += sdata[tid + 512];
  }

  __syncthreads();

  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] += sdata[tid + 256];
  }

  __syncthreads();

  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] += sdata[tid + 128];
  }

  __syncthreads();

  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] += sdata[tid + 64];
  }

  __syncthreads();

  if (tid < 32) warpReduce(sdata, tid);

  __syncthreads();//not needed?

  *g_odata = sdata[0];
  __syncthreads();


}

// z= a*z + x + b*y
__device__ void cudaDevicezaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=a*dz[row]  + dx[row] + b*dy[row];
}

// z= x*y
__device__ void cudaDevicemultxy(double* dz, double* dx,double* dy, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=dx[row]*dy[row];
}

// z= a*x + b*y
__device__ void cudaDevicezaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=a*dx[row] + b*dy[row];
}

// y= a*x + y
__device__ void cudaDeviceaxpy(double* dy,double* dx, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=a*dx[row] + dy[row];
}

// sqrt(sum ( (x_i*y_i)^2)/n)
__device__ void cudaDeviceVWRMS_Norm(double *g_idata1, double *g_idata2, double *g_odata, int n, int n_shr_empty)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  //unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;

  __syncthreads(); //Not needed (should)

/*
  double mySum = (i < n) ? g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i] : 0;
  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x]*g_idata2[i+blockDim.x];
*/

  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i];
  __syncthreads();

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] += sdata[tid + s];

    __syncthreads();
  }

  //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
  g_odata[0] = sqrt(sdata[0]/n);
  //*g_odata = sqrt(sdata[0]/n);
  __syncthreads();
}

// y=alpha*y
__device__ void cudaDevicescaley(double* dy, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=a*dy[row];
}

