#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
* Illinois at Urbana-Champaign
* SPDX-License-Identifier: MIT
*/

#include "libsolv.h"

__device__ void cudaDeviceSpmvCSR(double* dx, double* db, double* dA, int* djA, int* diA){
 __syncthreads();
 int i = threadIdx.x + blockDim.x*blockIdx.x;
 double sum = 0.0;
 int nnz=diA[blockDim.x];
 for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
   sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
 }
 __syncthreads();
 dx[i]=sum;
 __syncthreads();
}

__device__ void cudaDeviceSpmvCSC(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty){
 double mult;
 extern __shared__ double sdata[];
 int i= threadIdx.x + blockDim.x*blockIdx.x;
 unsigned int tid = threadIdx.x;
 __syncthreads();
 dx[i]=0.0;
 __syncthreads();
 int nnz=diA[blockDim.x];
 for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
   mult = db[i]*dA[j+nnz*blockIdx.x];
   atomicAdd_block(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
 }
 __syncthreads();
}

__device__ void cudaDeviceSpmvCSD(double* dx, double* db, double* dA, int* djA, int* diA){
 int tid = threadIdx.x + blockDim.x*blockIdx.x;
 __syncthreads();
 dx[tid]=0.0;
 __syncthreads();
 int nnz=1118;
 for(int iDiag=0; iDiag<blockDim.x; iDiag++) {
   if(threadIdx.x<diA[iDiag+1]-diA[iDiag]) {
     int dAi=diA[iDiag] + threadIdx.x  + nnz * blockIdx.x;
     int dbi=djA[diA[iDiag] + threadIdx.x] + blockDim.x*blockIdx.x;
     int dxi=((iDiag+djA[diA[iDiag] + threadIdx.x])%blockDim.x) + blockDim.x*blockIdx.x;
     dx[dxi] += db[dbi] * dA[dAi];
   }
   __syncthreads();
 }
}

__device__ void cudaDeviceSpmvBoolDet(double* dx, double* db, double* dA, int* diA)
{
 int tid = threadIdx.x + blockDim.x*blockIdx.x;
 __syncthreads();
 dx[tid]=0.0;
 __syncthreads();
}

__device__ void cudaDeviceSpmvCUID(double* dx, double* db, double* dA, int* djA)
{
 int tid = threadIdx.x + blockDim.x*blockIdx.x;
 __syncthreads();
 dx[tid]=0.0;
 __syncthreads();
 int nnz=1118;
 int iRow=threadIdx.x;
 __syncthreads();
 for(int row=0; row<blockDim.x; row++) {
   if (djA[threadIdx.x + row * blockDim.x] >= 0) {
     dx[iRow + blockDim.x * blockIdx.x] +=
         db[tid] * dA[djA[threadIdx.x + row * blockDim.x] + nnz * blockIdx.x];
#ifdef DEBUG_CUID
     printf("dx db dA djA %lf %lf %d\n", dx[iRow], db[tid],dA[djA[tid + row * blockDim.x]], djA[tid + row * blockDim.x]);
#endif
   }
   iRow++;
   if (iRow >= blockDim.x) {
     iRow = 0;
   }
   __syncthreads();
 }
}

__device__ void cudaDeviceSpmvCSRVector(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty)
{
 int t = threadIdx.x;
 int warpSize = 32;
 int lane = t & (warpSize-1);
 int warpsPerBlock = blockDim.x / warpSize;
 int row = (blockIdx.x * warpsPerBlock) + (t / warpSize);
 extern __shared__ double vals[];
 unsigned int tid = threadIdx.x;
 if(tid<n_shr_empty)
   vals[tid+blockDim.x]=0.;
 int rowStart = diA[row];
 int rowEnd = diA[row+1];
 double sum = 0.;
 for (int j = rowStart + lane; j < rowEnd; j += warpSize)
 {
   int col = djA[j];
   sum += dA[j] * db[col];
 }
 vals[t] = sum;
 __syncthreads();
 if (lane < 16) vals[t] += vals[t + 16];
 if (lane <  8) vals[t] += vals[t + 8];
 if (lane <  4) vals[t] += vals[t + 4];
 if (lane <  2) vals[t] += vals[t + 2];
 if (lane <  1) vals[t] += vals[t + 1];
 __syncthreads();
 if (lane == 0)
 {
   dx[row] = vals[t];
 }
}

#ifdef CSR_ADAPTIVE

__device__ int devicenextPowerOfTwo(int v) {
 v--;
 v |= v >> 1;
 v |= v >> 2;
 v |= v >> 4;
 v |= v >> 8;
 v |= v >> 16;
 v++;
 return v;
}

__device__ void cudaDevicedotxyCSRReduce(double *g_idata, double *g_idata2,
                                        double *g_odata, int n, int n_shr_empty, int n_shr_len)
{
 extern __shared__ double sdata[];
 unsigned int tid = threadIdx.x;
 unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
 __syncthreads();
 if(tid<n_shr_empty)
   sdata[tid+blockDim.x]=0.;

 __syncthreads();
 sdata[tid] = g_idata[i]*g_idata2[i];
 __syncthreads();

 for (unsigned int s=(n_shr_len)/2; s>0; s>>=1)
 {
   if (tid < s)
     sdata[tid] += sdata[tid + s];
   __syncthreads();
 }
 *g_odata = sdata[0];
 __syncthreads();
}

__device__ void cudaDeviceSpmvCSRReduce(double* dx, double* db, int nrows, double* dA, int* djA, int* diA){
 __syncthreads();
 int row = threadIdx.x + blockDim.x*blockIdx.x;
 double sum = 0.0;
 int nnz=diA[blockDim.x];
 int n_iters = nnz / blockDim.x; //todo /2?
 for(int i=0; i<n_iters; i++)
 {
   int offsetdA=diA[threadIdx.x+1]-diA[threadIdx.x];
   int n_shr_len=devicenextPowerOfTwo(offsetdA);
   int n_shr_empty=n_shr_len-(offsetdA);
   int j=row;
   dx[row] = db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
   int idx=threadIdx.x/offsetdA;
   cudaDevicedotxyCSRReduce(&db[djA[j]+blockDim.x*blockIdx.x],
                            &dA[j+nnz*blockIdx.x],&dx[idx],n_shr_empty,n_shr_len);
 }
 __syncthreads();
 int residual=nnz-(blockDim.x*n_iters);
 if(threadIdx.x < residual){
 }
}

#endif

__device__ void cudaDeviceSpmv(double* dx, double* db, double* dA, int* djA, int* diA, int n_shr_empty)
{
#ifdef CSR
 cudaDeviceSpmvCSR(dx,db,dA,djA,diA);
#elif CSC
 cudaDeviceSpmvCSC(dx,db,dA,djA,diA,n_shr_empty);
#elif CSD
 cudaDeviceSpmvCSD(dx,db,dA,djA,diA);
#elif CBD
 cudaDeviceSpmvBoolDet(dx,db,dA,djA);
#elif CUID
 cudaDeviceSpmvCUID(dx,db,dA,djA);
#elif CSR_VECTOR
 cudaDeviceSpmvCSRVector(dx,db,dA,djA,diA);
#elif CSR_ADAPTIVE
 cudaDeviceSpmvCSRReduce(dx,db,dA,djA,diA);
#else
 cudaDeviceSpmvCSR(dx,db,dA,djA,diA);
#endif
}

__device__ void warpReduce_2(volatile double *sdata, unsigned int tid) {
 unsigned int blockSize = blockDim.x;
 if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
 if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
 if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
 if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
 if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
 if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__device__ void cudaDevicedotxy(double *g_idata1, double *g_idata2,
                               double *g_odata, int n_shr_empty){
 extern __shared__ double sdata[];
 unsigned int tid = threadIdx.x;
 __syncthreads();
 if(tid<n_shr_empty)
   sdata[tid+blockDim.x]=0.;
 __syncthreads();
 //print_double(sdata,73,"sdata");
#ifdef DEV_cudaDevicedotxy_2
 //used for compare with cpu
 sdata[0]=0.;
 __syncthreads();
 if(tid==0){
   for(int j=0;j<blockDim.x;j++){
     sdata[0]+=g_idata1[j+blockIdx.x*blockDim.x]*g_idata2[j+blockIdx.x*blockDim.x];
   }
 }
#else
 unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
 if(tid<n_shr_empty)
   sdata[tid+blockDim.x]=0.;
 __syncthreads();
 sdata[tid] = g_idata1[i]*g_idata2[i];
 __syncthreads();
 unsigned int blockSize = blockDim.x+n_shr_empty;
 if ((blockSize >= 1024) && (tid < 512)) {
   sdata[tid] += sdata[tid + 512];
 }
 __syncthreads();
 if ((blockSize >= 512) && (tid < 256)) {
   sdata[tid] += sdata[tid + 256];
 }
 __syncthreads();
 if ((blockSize >= 256) && (tid < 128)) {
   sdata[tid] += sdata[tid + 128];
 }
 __syncthreads();
 if ((blockSize >= 128) && (tid < 64)) {
   sdata[tid] += sdata[tid + 64];
 }
 __syncthreads();
 if (tid < 32) warpReduce_2(sdata, tid);
#endif
 __syncthreads();
 *g_odata = sdata[0];
 __syncthreads();
}

__device__
   void solveBcgCudaDeviceCVODE(ModelDataGPU *md){
 int i = blockIdx.x * blockDim.x + threadIdx.x;
 double alpha,rho0,omega0,beta,rho1,temp1,temp2;
 alpha=rho0=omega0=beta=rho1=temp1=temp2=1.0;
 md->dn0[i]=0.0;
 md->dp0[i]=0.0;
 cudaDeviceSpmv(md->dr0,md->dx,md->dA,md->djA,md->diA, md->n_shr_empty);
 md->dr0[i]=md->dtempv[i]-md->dr0[i];
 md->dr0h[i]=md->dr0[i];
 int it=0;
 while(it<1000 && temp1>1.0E-30){
   cudaDevicedotxy(md->dr0, md->dr0h, &rho1, md->n_shr_empty);
   beta = (rho1 / rho0) * (alpha / omega0);
   md->dp0[i]=beta*md->dp0[i]+md->dr0[i]-md->dn0[i]*omega0*beta;
   md->dy[i]=md->ddiag[i]*md->dp0[i];
   cudaDeviceSpmv(md->dn0, md->dy, md->dA, md->djA, md->diA, md->n_shr_empty);
   cudaDevicedotxy(md->dr0h, md->dn0, &temp1, md->n_shr_empty);
   alpha = rho1 / temp1;
   md->ds[i]=md->dr0[i]-alpha*md->dn0[i];
   md->dx[i]+=alpha*md->dy[i];
   md->dy[i]=md->ddiag[i]*md->ds[i];
   cudaDeviceSpmv(md->dt, md->dy, md->dA, md->djA, md->diA, md->n_shr_empty);
   md->dr0[i]=md->ddiag[i]*md->dt[i];
   cudaDevicedotxy(md->dy, md->dr0, &temp1, md->n_shr_empty);
   cudaDevicedotxy(md->dr0, md->dr0, &temp2, md->n_shr_empty);
   omega0 = temp1 / temp2;
   md->dx[i]+=omega0*md->dy[i];
   md->dr0[i]=md->ds[i]-omega0*md->dt[i];
   md->dt[i]=0.0;
   cudaDevicedotxy(md->dr0, md->dr0, &temp1, md->n_shr_empty);
   temp1 = sqrt(temp1);
   rho0 = rho1;
   it++;
   __syncthreads();
 }
}

__global__
   void cudaGlobalCVode(ModelDataGPU md_object) {
 ModelDataGPU *md = &md_object;
 int i = blockIdx.x * blockDim.x + threadIdx.x;
 if(i<md->nrows) {
   solveBcgCudaDeviceCVODE(md);
 }
}

int nextPowerOfTwoBCG(int v) {
 v--;
 v |= v >> 1;
 v |= v >> 2;
 v |= v >> 4;
 v |= v >> 8;
 v |= v >> 16;
 v++;
 return v;
}

void solveGPU_block(ModelDataGPU* mGPU){
 int len_cell = mGPU->nrows / mGPU->n_cells;
 int threads_block = len_cell;
 int blocks = mGPU->n_cells;
 int n_shr_memory = nextPowerOfTwoBCG(len_cell);
 mGPU->n_shr_empty = n_shr_memory - threads_block;
 cudaGlobalCVode<<< blocks, threads_block,
                   n_shr_memory * sizeof(double)>>>(*mGPU);
}
